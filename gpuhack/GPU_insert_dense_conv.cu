#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <array>
#include <iostream>
#include <cassert>
#include <limits>
#include <chrono>
#include <iomanip>

#include "data_structures/APR/APR.hpp"
#include "data_structures/APR/APRTreeIterator.hpp"
#include "data_structures/APR/ExtraParticleData.hpp"
#include "data_structures/Mesh/MeshData.hpp"
#include "io/TiffUtils.hpp"

#include "thrust/device_vector.h"
#include "thrust/tuple.h"
#include "thrust/copy.h"


struct cmdLineOptions{
    std::string output = "output";
    std::string stats = "";
    std::string directory = "";
    std::string input = "";
};

bool command_option_exists(char **begin, char **end, const std::string &option) {
    return std::find(begin, end, option) != end;
}

char* get_command_option(char **begin, char **end, const std::string &option) {
    char ** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end) {
        return *itr;
    }
    return 0;
}

cmdLineOptions read_command_line_options(int argc, char **argv) {
    cmdLineOptions result;

    if(argc == 1) {
        std::cerr << "Usage: \"Example_apr_neighbour_access -i input_apr_file -d directory\"" << std::endl;
        exit(1);
    }
    if(command_option_exists(argv, argv + argc, "-i")) {
        result.input = std::string(get_command_option(argv, argv + argc, "-i"));
    } else {
        std::cout << "Input file required" << std::endl;
        exit(2);
    }
    if(command_option_exists(argv, argv + argc, "-d")) {
        result.directory = std::string(get_command_option(argv, argv + argc, "-d"));
    }
    if(command_option_exists(argv, argv + argc, "-o")) {
        result.output = std::string(get_command_option(argv, argv + argc, "-o"));
    }

    return result;
}


void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half);


__global__ void insert(
    std::size_t _level,
    std::size_t _z_index,
    const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
    const std::uint16_t*           _y_ex,
    const std::uint16_t*           _pdata,
    const std::size_t*             _offsets,
    std::size_t                    _max_y,
    std::size_t                    _max_x,
    std::size_t                    _nparticles,
    std::uint16_t*                 _temp_vec,
    std::size_t                    _stencil_size,
    std::size_t                    _stencil_half
    ){

    unsigned int x_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(x_index >= _max_x){
        return; // out of bounds
    }

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index + x_index;
    auto row_start = _line_offsets[level_zx_offset];


    auto particle_index_begin = thrust::get<0>(row_start);
    auto particle_index_end   = thrust::get<1>(row_start);

    //_z_index = 0;
    auto t_index = x_index*_max_y + ((_z_index % _stencil_size)*_max_y*_max_x) ;

    for(auto y = 0;y < _max_y;y++){
        _temp_vec[t_index+y]  = 0;
    }

    if(thrust::get<1>(row_start) == 0)
        return;


    for (std::size_t global_index = particle_index_begin;
         global_index <= particle_index_end; ++global_index) {

        uint16_t current_particle_value = _pdata[global_index];
        auto y = _y_ex[global_index];
        _temp_vec[t_index+y] = current_particle_value;

    }

}

__global__ void push_back(
    std::size_t _level,
    std::size_t _z_index,
    const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
    const std::uint16_t*           _y_ex,
    const std::uint16_t*           _temp_vec,
    const std::size_t*             _offsets,
    std::size_t                    _max_y,
    std::size_t                    _max_x,
    std::size_t 		   _max_z,
    std::size_t                    _nparticles,
    std::uint16_t*                 _pdata,
    std::size_t                    _stencil_size,
    std::size_t                    _stencil_half,
    const std::float_t*           _stencil
    ){

    unsigned int x_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(x_index >= _max_x){
        return; // out of bounds
    }

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index + x_index;
    auto row_start = _line_offsets[level_zx_offset];

    if(thrust::get<1>(row_start) == 0)
        return;

    auto particle_index_begin = thrust::get<0>(row_start);
    auto particle_index_end   = thrust::get<1>(row_start);

    //_z_index=0;
    auto t_index = x_index*_max_y + ((_z_index % _stencil_size)*_max_y*_max_x) ;
    auto temp_index = 0;

    // Convolution begin 
    for (std::size_t global_index = particle_index_begin;
         global_index <= particle_index_end; ++global_index) {

	    int counter = 0;
	    double neighbour_sum = 0;
        auto y = _y_ex[global_index];

        //_pdata[global_index]=1;
        _pdata[global_index]=0;


        int lower_bound = (_stencil_half);

        for(int q = -(lower_bound); q < (lower_bound+1); ++q){	 // z stencil
            for(int l = -(lower_bound); l < (lower_bound+1); ++l){   // x stencil
                for(int w = -(lower_bound); w < (lower_bound+1); ++w){	// y stencil

                    if((x_index + l) >= 0 && (x_index + l) < _max_x){
                        if((_z_index + q) >= 0 && (_z_index + q) < _max_z){
                            if((y + w) >= 0 && (y + w) < _max_y){

                                temp_index = (x_index + l)*_max_y + (((_z_index+q+ _stencil_size) % _stencil_size)*_max_y*_max_x) +y+w ;
                                neighbour_sum += (_stencil[counter]*_temp_vec[temp_index]);
                                //
                                //if(q==1) {
                                //    temp_index = (x_index)*_max_y + (((_z_index+q+ _stencil_size) % _stencil_size)*_max_y*_max_x) +y ;
                                  //  _pdata[global_index] = _temp_vec[temp_index];
                                //}
                                counter++;

                             }
                        }
                    }


                }
            }
        }

        _pdata[global_index] = std::round(neighbour_sum/27.0);


       // temp_index = (x_index)*_max_y + (((_z_index) % _stencil_size)*_max_y*_max_x) ;
       // _pdata[global_index] = _temp_vec[temp_index+y];

    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    // Read provided APR file
    cmdLineOptions options = read_command_line_options(argc, argv);
    const int reps = 20;

    std::string fileName = options.directory + options.input;
    APR<uint16_t> apr;
    apr.read_apr(fileName);

    // Get dense representation of APR
    APRIterator<uint16_t> aprIt(apr);

    ///////////////////////////
    ///
    /// Sparse Data for GPU
    ///
    ///////////////////////////

    std::vector<std::tuple<std::size_t,std::size_t>> level_zx_index_start;//size = number of rows on all levels
    std::vector<std::uint16_t> y_explicit;y_explicit.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::uint16_t> particle_values;particle_values.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::size_t> level_offset(aprIt.level_max()+1,UINT64_MAX);//size = number of levels
    const int stencil_half = 2;
    const int stencil_size = 2*stencil_half+1; 
    std::vector<std::float_t> stencil;		// the stencil on the host
    std::float_t stencil_value = 1;
    stencil.resize(pow(stencil_half*2 + 1,stencil_size),stencil_value);

    std::cout << stencil[0] << std::endl;


    std::size_t x = 0;
    std::size_t z = 0;

    std::size_t zx_counter = 0;


    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        level_offset[level] = zx_counter;

        for (z = 0; z < aprIt.spatial_index_z_max(level); ++z) {
            for (x = 0; x < aprIt.spatial_index_x_max(level); ++x) {

                zx_counter++;
                if (aprIt.set_new_lzx(level, z, x) < UINT64_MAX) {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>(aprIt.global_index(),
                                                                                               aprIt.particles_zx_end(level,z,x)-1)); //This stores the begining and end global index for each level_xz_row
                } else {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>(UINT64_MAX, 0)); //This stores the begining and end global index for each level_
                }

                for (aprIt.set_new_lzx(level, z, x);
                     aprIt.global_index() < aprIt.particles_zx_end(level, z,
                                                                   x); aprIt.set_iterator_to_particle_next_particle()) {
                    y_explicit.emplace_back(aprIt.y());
                    particle_values.emplace_back(apr.particles_intensities[aprIt]);

                }
            }

        }
    }

    // std::vector<uint16_t> cpu_access_data(apr.particles_intensities.data.size(),std::numeric_limits<std::uint16_t>::max());

    // for ( int r = 0;r<reps;++r){
    //     auto start_cpu = std::chrono::high_resolution_clock::now();


    //     for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {

    //         const int x_num = aprIt.spatial_index_x_max(level);
    //         //const int z_num = aprIt.spatial_index_z_max(level);

    //         for (z = 0; z < aprIt.spatial_index_z_max(level); ++z) {
    //             for (x = 0; x < aprIt.spatial_index_x_max(level); ++x) {
    //                 if(level_offset[level]<UINT64_MAX) {
    //                     uint64_t level_xz_offset = level_offset[level] + x_num * z + x;
    //                     if (std::get<1>(level_zx_index_start[level_xz_offset])) {
    //                         uint64_t particle_index_begin = std::get<0>(level_zx_index_start[level_xz_offset]);
    //                         uint64_t particle_index_end = std::get<1>(level_zx_index_start[level_xz_offset]);

    //                         for (uint64_t global_index = particle_index_begin;
    //                              global_index <= particle_index_end; ++global_index) {

    //                             uint16_t current_particle_value = particle_values[global_index];

    //                             cpu_access_data[global_index] = (current_particle_value);

    //                         }
    //                     }
    //                 }

    //             }
    //         }
    //     }

    //     auto end_cpu = std::chrono::high_resolution_clock::now();

    //     std::chrono::duration<double, std::milli> diff_cpu = end_cpu-start_cpu;
    //     std::cout << std::setw(3) << r << " CPU:      " << diff_cpu   .count() << " ms\n";

    // }

    ////////////////////
    ///
    /// Example of doing our level,z,x access using the GPU data structure
    ///
    /////////////////////
    auto start = std::chrono::high_resolution_clock::now();


    thrust::host_vector<thrust::tuple<std::size_t,std::size_t> > h_level_zx_index_start(level_zx_index_start.size());
    thrust::transform(level_zx_index_start.begin(), level_zx_index_start.end(),
                      h_level_zx_index_start.begin(),
                      [] ( const auto& _el ){
                          return thrust::make_tuple(std::get<0>(_el), std::get<1>(_el));
                      } );

    thrust::device_vector<thrust::tuple<std::size_t,std::size_t> > d_level_zx_index_start = h_level_zx_index_start;


    thrust::device_vector<std::float_t> d_stencil(stencil.begin(), stencil.end());		// device stencil
    thrust::device_vector<std::uint16_t> d_y_explicit(y_explicit.begin(), y_explicit.end());
    thrust::device_vector<std::uint16_t> d_particle_values(particle_values.begin(), particle_values.end());
    thrust::device_vector<std::uint16_t> d_test_access_data(d_particle_values.size(),0);

    thrust::device_vector<std::size_t> d_level_offset(level_offset.begin(),level_offset.end());

    std::size_t max_elements = 0;
 
    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        auto xtimesy = aprIt.spatial_index_y_max(level);// + (stencil_size - 1);
        xtimesy *= aprIt.spatial_index_x_max(level);// + (stencil_size - 1);
        if(max_elements < xtimesy)
            max_elements = xtimesy;
    }
    thrust::device_vector<std::uint16_t> d_temp_vec(max_elements*stencil_size,0);

    const thrust::tuple<std::size_t,std::size_t>* levels =  thrust::raw_pointer_cast(d_level_zx_index_start.data());
    const std::uint16_t*             y_ex   =  thrust::raw_pointer_cast(d_y_explicit.data());
    const std::uint16_t*             pdata  =  thrust::raw_pointer_cast(d_particle_values.data());
    const std::size_t*             offsets= thrust::raw_pointer_cast(d_level_offset.data());
    std::uint16_t*                   tvec = thrust::raw_pointer_cast(d_temp_vec.data());
    std::uint16_t*                   expected = thrust::raw_pointer_cast(d_test_access_data.data());
    const std::float_t*		     stencil_pointer =  thrust::raw_pointer_cast(d_stencil.data());		// stencil pointer

     if(hipGetLastError()!=hipSuccess){
        std::cerr << "memory transfers failed!\n";
    }
    auto end_gpu_tx = std::chrono::high_resolution_clock::now();

    for ( int r = 0;r<reps;++r){

        auto start_gpu_kernel = std::chrono::high_resolution_clock::now();

        for (int lvl = aprIt.level_min(); lvl <= aprIt.level_max(); ++lvl) {

            const int y_num = aprIt.spatial_index_y_max(lvl);
            const int x_num = aprIt.spatial_index_x_max(lvl);
            const int z_num = aprIt.spatial_index_z_max(lvl);

            dim3 threads(32);
            dim3 blocks((x_num + threads.x- 1)/threads.x);

            for (int i = 0; i < stencil_half; ++i) {
                insert<<<blocks,threads>>>(lvl,
                        i,
                        levels,
                        y_ex,
                        pdata,
                        offsets,
                        y_num,x_num,
                        particle_values.size(),
                        tvec,
                        stencil_size, stencil_half);
            }


            for(int z = 0;z<(z_num-stencil_half);++z){

                insert<<<blocks,threads>>>(lvl,
                                           z+stencil_half,
                                           levels,
                                           y_ex,
                                           pdata,
                                           offsets,
                                           y_num,x_num,
                                           particle_values.size(),
                                           tvec,
                                           stencil_size, stencil_half);

                if(hipGetLastError()!=hipSuccess){
                    std::cerr << "on " << lvl << " the cuda kernel does not run!\n";
                    break;
                }
                hipDeviceSynchronize();

                push_back<<<blocks,threads>>>(lvl,
                                              z,
                                              levels,
                                              y_ex,
                                              tvec,
                                              offsets,
                                              y_num,x_num,z_num,
                                              particle_values.size(),
                                              expected,
                                              stencil_size, stencil_half, stencil_pointer);

                hipDeviceSynchronize();
            }
        }
        hipDeviceSynchronize();
        auto end_gpu_kernel = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double,std::milli> rep_diff = end_gpu_kernel - start_gpu_kernel;
        std::cout << std::setw(3) << r << " GPU:      " << rep_diff  .count() << " ms\n";

    }

    auto end_gpu_kernels = std::chrono::high_resolution_clock::now();

    std::vector<std::uint16_t> test_access_data(d_test_access_data.size(),std::numeric_limits<std::uint16_t>::max());
    thrust::copy(d_test_access_data.begin(), d_test_access_data.end(), test_access_data.begin());

    auto end_gpu = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double,std::milli> gpu_tx_up = end_gpu_tx - start;
    std::chrono::duration<double,std::milli> gpu_tx_down = end_gpu - end_gpu_kernels;

    std::cout << "   GPU: up   " << gpu_tx_up  .count() << " ms\n";
    std::cout << "   GPU: down " << gpu_tx_down.count() << " ms\n";

   // assert(test_access_data.back() != std::numeric_limits<std::uint16_t>::max());

    //////////////////////////
    ///
    /// Now check the data
    ///
    ////////////////////////////

    ExtraParticleData<float> utest_data(apr);
    apr.parameters.input_dir = options.directory;


    create_test_particles_surya(apr,aprIt, utest_data,apr.particles_intensities,stencil, stencil_size, stencil_half);

    bool success = true;

    uint64_t c_fail= 0;



    for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
        //This step is required for all loops to set the iterator by the particle number
        aprIt.set_iterator_to_particle_by_number(particle_number);
        if(utest_data.data[particle_number]!=test_access_data[particle_number]){
            success = false;
            std::cout << aprIt.x()<< " "  << aprIt.y()<< " "  << aprIt.z() << " "<< aprIt.level() << " expected: " << utest_data.data[particle_number] << ", received: " << test_access_data[particle_number] << "\n";
            //break;
            c_fail++;
        }

       // std::cout << aprIt.x()<< " "  << aprIt.y()<< " "  << aprIt.z() << " "<< aprIt.level() << " expected: " << utest_data.data[particle_number] << ", received: " << test_access_data[particle_number] << "\n";

    }


    if(success){
        std::cout << "PASS" << std::endl;
    } else {
        std::cout << "FAIL " << c_fail << std::endl;
    }


}



void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half){

    for (uint64_t level_local = apr_iterator.level_max(); level_local >= apr_iterator.level_min(); --level_local) {


        MeshData<float> by_level_recon;
        by_level_recon.init(apr_iterator.spatial_index_y_max(level_local),apr_iterator.spatial_index_x_max(level_local),apr_iterator.spatial_index_z_max(level_local),0);

        uint64_t level = level_local;

            const int step_size = 1;

            uint64_t particle_number;

            for (particle_number = apr_iterator.particles_level_begin(level);
                 particle_number < apr_iterator.particles_level_end(level); ++particle_number) {
                //
                //  Parallel loop over level
                //
                apr_iterator.set_iterator_to_particle_by_number(particle_number);

                int dim1 = apr_iterator.y() ;
                int dim2 = apr_iterator.x() ;
                int dim3 = apr_iterator.z() ;

                float temp_int;
                //add to all the required rays

                temp_int = particles[apr_iterator];

                const int offset_max_dim1 = std::min((int) by_level_recon.y_num, (int) (dim1 + step_size));
                const int offset_max_dim2 = std::min((int) by_level_recon.x_num, (int) (dim2 + step_size));
                const int offset_max_dim3 = std::min((int) by_level_recon.z_num, (int) (dim3 + step_size));

                for (int64_t q = dim3; q < offset_max_dim3; ++q) {

                    for (int64_t k = dim2; k < offset_max_dim2; ++k) {
                        for (int64_t i = dim1; i < offset_max_dim1; ++i) {
                            by_level_recon.mesh[i + (k) * by_level_recon.y_num + q * by_level_recon.y_num * by_level_recon.x_num] = temp_int;
                        }
                    }
                }
            }


        int x = 0;
        int z = 0;


        for (z = 0; z < (apr.spatial_index_z_max(level)-stencil_half); ++z) {
            //lastly loop over particle locations and compute filter.
            for (x = 0; x < apr.spatial_index_x_max(level); ++x) {
                for (apr_iterator.set_new_lzx(level, z, x);
                     apr_iterator.global_index() < apr_iterator.particles_zx_end(level, z,
                                                                                 x); apr_iterator.set_iterator_to_particle_next_particle()) {
                    double neigh_sum = 0;
                    float counter = 0;

                    const int k = apr_iterator.y(); // offset to allow for boundary padding
                    const int i = x;

                    //test_particles[apr_iterator]=0;

                    for (int l = -stencil_half; l < stencil_half+1; ++l) {
                        for (int q = -stencil_half; q < stencil_half+1; ++q) {
                            for (int w = -stencil_half; w < stencil_half+1; ++w) {

                                if((k+w)>=0 & (k+w) < (apr.spatial_index_y_max(level))){
                                    if((i+q)>=0 & (i+q) < (apr.spatial_index_x_max(level))){
                                        if((z+l)>=0 & (z+l) < (apr.spatial_index_z_max(level))){
                                            neigh_sum += stencil[counter] * by_level_recon.at(k + w, i + q, z+l);
                                            //neigh_sum += by_level_recon.at(k + w, i + q, z+l);
                                            //if(l==1) {
                                              //  test_particles[apr_iterator] = by_level_recon.at(k, i , z+l);
                                            //}
                                        }
                                    }
                                }
               		            counter++;
                            }
                        }
                    }

                    test_particles[apr_iterator] = std::round(neigh_sum/27.0);


                }
            }
        }

         std::string image_file_name = apr.parameters.input_dir + std::to_string(level_local) + "_by_level.tif";
        TiffUtils::saveMeshAsTiff(image_file_name, by_level_recon);
    }

}

