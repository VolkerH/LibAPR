#include "hip/hip_runtime.h"
//
// Created by cheesema on 09.03.18.
//
#include <algorithm>
#include <vector>
#include <array>
#include <iostream>
#include <cassert>
#include <limits>
#include <chrono>
#include <iomanip>

#include "data_structures/APR/APR.hpp"
#include "data_structures/APR/APRTreeIterator.hpp"
#include "data_structures/APR/ExtraParticleData.hpp"
#include "data_structures/Mesh/MeshData.hpp"
#include "io/TiffUtils.hpp"

#include "thrust/device_vector.h"
#include "thrust/tuple.h"
#include "thrust/copy.h"


struct cmdLineOptions{
    std::string output = "output";
    std::string stats = "";
    std::string directory = "";
    std::string input = "";
};

bool command_option_exists(char **begin, char **end, const std::string &option) {
    return std::find(begin, end, option) != end;
}

char* get_command_option(char **begin, char **end, const std::string &option) {
    char ** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end) {
        return *itr;
    }
    return 0;
}

cmdLineOptions read_command_line_options(int argc, char **argv) {
    cmdLineOptions result;

    if(argc == 1) {
        std::cerr << "Usage: \"Example_apr_neighbour_access -i input_apr_file -d directory\"" << std::endl;
        exit(1);
    }
    if(command_option_exists(argv, argv + argc, "-i")) {
        result.input = std::string(get_command_option(argv, argv + argc, "-i"));
    } else {
        std::cout << "Input file required" << std::endl;
        exit(2);
    }
    if(command_option_exists(argv, argv + argc, "-d")) {
        result.directory = std::string(get_command_option(argv, argv + argc, "-d"));
    }
    if(command_option_exists(argv, argv + argc, "-o")) {
        result.output = std::string(get_command_option(argv, argv + argc, "-o"));
    }

    return result;
}


void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half);


__global__ void insert(
        std::size_t _level,
        std::size_t _z_index,
        const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
        const std::uint16_t*           _y_ex,
        const std::uint16_t*           _pdata,
        const std::size_t*             _offsets,
        std::size_t                    _max_y,
        std::size_t                    _max_x,
        std::size_t                    _nparticles,
        std::uint16_t*                 _temp_vec,
        std::size_t                    _stencil_size,
        std::size_t                    _stencil_half
){

    unsigned int x_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(x_index >= _max_x){
        return; // out of bounds
    }

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index + x_index;
    auto row_start = _line_offsets[level_zx_offset];


    auto particle_index_begin = thrust::get<0>(row_start);
    auto particle_index_end   = thrust::get<1>(row_start);

    //_z_index = 0;
    auto t_index = x_index*_max_y + ((_z_index % _stencil_size)*_max_y*_max_x) ;

//    for(auto y = 0;y < _max_y;y++){
//        _temp_vec[t_index+y]  = 0;
//    }

    if((particle_index_end-particle_index_begin) == 0)
        return;


    for (std::size_t global_index = particle_index_begin;
         global_index < particle_index_end; ++global_index) {

        uint16_t current_particle_value = _pdata[global_index];
        auto y = _y_ex[global_index];
        _temp_vec[t_index+y] = current_particle_value;

    }

}

__global__ void insert_dynamic(
        std::size_t _level,
        std::size_t _z_index,
        const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
        const std::uint16_t*           _y_ex,
        const std::uint16_t*           _pdata,
        const std::size_t*             _offsets,
        std::size_t                    _max_y,
        std::size_t                    _max_x,
        std::size_t                    _nparticles,
        std::uint16_t*                 _temp_vec,
        std::size_t                    _stencil_size,
        std::size_t                    _stencil_half,std::uint16_t*  _xend,std::size_t*  _ind_end,std::size_t num_blocks
){


    int load_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(load_index>=num_blocks){
        return;
    }
    //_z_index = 0;



    std::size_t x_begin;
    std::size_t x_end = _xend[load_index];

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index;

    std::size_t parts_begin;
    std::size_t parts_end;

    if(load_index==0){
        x_begin = 0;
        parts_begin = thrust::get<0>(_line_offsets[level_zx_offset]);

    } else {
        x_begin = _xend[load_index-1];
        parts_begin = _ind_end[load_index-1];
    }

    //

    for (auto x = x_begin; x <= x_end; ++x) {


        auto t_index = x*_max_y + ((_z_index % _stencil_size)*_max_y*_max_x) ;

        if(x!=x_begin){
            parts_begin  = thrust::get<0>(_line_offsets[level_zx_offset+x]);
        }

        if(x!=x_end){
            parts_end  = thrust::get<1>(_line_offsets[level_zx_offset+x]);
        } else {
            parts_end  = _ind_end[load_index];
        }


        for (std::size_t p = parts_begin; p < parts_end ; ++p) {
            //uint16_t current_particle_value = ;
            auto y = _y_ex[p];
            _temp_vec[t_index+y] = _pdata[p];
        }

    }


}


__global__ void load_balance(std::size_t _level,std::size_t _z_index,const thrust::tuple<std::size_t,std::size_t>* _line_offsets,std::uint16_t*  _xend,std::size_t*  _ind_end,const std::size_t* _offsets,
                             std::size_t   _max_x,std::size_t num_blocks,std::float_t parts_per_block,std::size_t parts_begin){

    int x_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(x_index >= _max_x){
        return; // out of bounds
    }

    //printf("Hello from dim: %d block: %d, thread: %d  x index: %d z: %d \n",blockDim.x, blockIdx.x, threadIdx.x,x_index,(int) _z_index);

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index + x_index;

    std::size_t parts_end = thrust::get<1>(_line_offsets[level_zx_offset]);

    std::size_t index_begin =  floor((thrust::get<0>(_line_offsets[level_zx_offset])-parts_begin)/parts_per_block);

    std::size_t index_end;

    if(parts_end==parts_begin){
        index_end=0;
    } else {
        index_end = floor((parts_end-parts_begin)/parts_per_block);
    }

    //need to add the loop
    if(index_begin!=index_end){

        if(index_begin==0){
            _ind_end[0] = parts_begin;
            _xend[0] = 0;
        }

        for (int i = (index_begin+1); i <= index_end; ++i) {
            _xend[i]=x_index;
            _ind_end[i]= floor(i*parts_per_block) + parts_begin;

        }
    }



    if(x_index==(_max_x-1)){
        _ind_end[num_blocks-1] = parts_end;
        _xend[num_blocks-1] = (_max_x-1);

    }



}


__global__ void test_iterate(std::size_t _level,std::size_t _z_index,const thrust::tuple<std::size_t,std::size_t>* _line_offsets,std::uint16_t*  _xend,std::size_t*  _ind_end,const std::size_t* _offsets,
        std::size_t   _max_x,std::uint16_t*  _pdata,std::size_t num_blocks){

    int load_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(load_index>=num_blocks){
        return;
    }

    std::size_t x_begin;
    std::size_t x_end = _xend[load_index];

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index;

    std::size_t parts_begin;
    std::size_t parts_end;

    if(load_index==0){
        x_begin = 0;
        parts_begin = thrust::get<0>(_line_offsets[level_zx_offset]);

    } else {
        x_begin = _xend[load_index-1];
        parts_begin = _ind_end[load_index-1];
    }

    //

    for (auto x = x_begin; x <= x_end; ++x) {


        if(x!=x_begin){
            parts_begin  = thrust::get<0>(_line_offsets[level_zx_offset+x]);
        }

        if(x!=x_end){
            parts_end  = thrust::get<1>(_line_offsets[level_zx_offset+x]);
        } else {
            parts_end  = _ind_end[load_index];
        }


         for (std::size_t p = parts_begin; p < parts_end ; ++p) {
           _pdata[p]+=1;
         }

    }

}


__global__ void push_back_dynamic(
        std::size_t _level,
        std::size_t _z_index,
        const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
        const std::uint16_t*           _y_ex,
        const std::uint16_t*           _temp_vec,
        const std::size_t*             _offsets,
        std::size_t                    _max_y,
        std::size_t                    _max_x,
        std::size_t 		   _max_z,
        std::size_t                    _nparticles,
        std::uint16_t*                 _pdata,
        std::size_t                    _stencil_size,
        std::size_t                    _stencil_half,
        const std::float_t*           _stencil,std::uint16_t*  _xend,std::size_t*  _ind_end,std::size_t num_blocks
){


    int load_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(load_index>=num_blocks){
        return;
    }
    //_z_index = 0;



    std::size_t x_begin;
    std::size_t x_end = _xend[load_index];

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index;

    std::size_t parts_begin;
    std::size_t parts_end;

    if(load_index==0){
        x_begin = 0;
        parts_begin = thrust::get<0>(_line_offsets[level_zx_offset]);

    } else {
        x_begin = _xend[load_index-1];
        parts_begin = _ind_end[load_index-1];
    }

    //
    std::size_t temp_index;

    for (auto x = x_begin; x <= x_end; ++x) {


        auto t_index = x*_max_y + ((_z_index % _stencil_size)*_max_y*_max_x) ;

        if(x!=x_begin){
            parts_begin  = thrust::get<0>(_line_offsets[level_zx_offset+x]);
        }

        if(x!=x_end){
            parts_end  = thrust::get<1>(_line_offsets[level_zx_offset+x]);
        } else {
            parts_end  = _ind_end[load_index];
        }


        for (std::size_t global_index = parts_begin; global_index < parts_end ; ++global_index) {

            int counter = 0;
            double neighbour_sum = 0;
            auto y = _y_ex[global_index];

            int lower_bound = (_stencil_half);

            for(int q = -(lower_bound); q < (lower_bound+1); ++q){	 // z stencil
                for(int l = -(lower_bound); l < (lower_bound+1); ++l){   // x stencil
                    for(int w = -(lower_bound); w < (lower_bound+1); ++w){	// y stencil

                        if((x + l) >= 0 && (x + l) < _max_x){
                            if((_z_index + q) >= 0 && (_z_index + q) < _max_z){
                                if((y + w) >= 0 && (y + w) < _max_y){

                                    temp_index = (x + l)*_max_y + (((_z_index+q+ _stencil_size) % _stencil_size)*_max_y*_max_x) +y+w ;
                                    neighbour_sum += (_stencil[counter]*_temp_vec[temp_index]);
                                    //
                                    counter++;

                                }
                            }
                        }


                    }
                }
            }

            _pdata[global_index] = std::round(neighbour_sum/(pow(_stencil_size,3)*1.0));


        }

    }

}



__global__ void push_back(
        std::size_t _level,
        std::size_t _z_index,
        const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
        const std::uint16_t*           _y_ex,
        const std::uint16_t*           _temp_vec,
        const std::size_t*             _offsets,
        std::size_t                    _max_y,
        std::size_t                    _max_x,
        std::size_t 		   _max_z,
        std::size_t                    _nparticles,
        std::uint16_t*                 _pdata,
        std::size_t                    _stencil_size,
        std::size_t                    _stencil_half,
        const std::float_t*           _stencil
){

    unsigned int x_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(x_index >= _max_x){
        return; // out of bounds
    }

    auto level_zx_offset = _offsets[_level] + _max_x * _z_index + x_index;
    auto row_start = _line_offsets[level_zx_offset];



    auto particle_index_begin = thrust::get<0>(row_start);
    auto particle_index_end   = thrust::get<1>(row_start);

    if((particle_index_end-particle_index_begin) == 0)
        return;

    //_z_index=0;
    auto t_index = x_index*_max_y + ((_z_index % _stencil_size)*_max_y*_max_x) ;
    auto temp_index = 0;

    // Convolution begin
    for (std::size_t global_index = particle_index_begin;
         global_index < particle_index_end; ++global_index) {

        int counter = 0;
        double neighbour_sum = 0;
        auto y = _y_ex[global_index];

        //_pdata[global_index]=1;
        _pdata[global_index]=0;


        int lower_bound = (_stencil_half);

        for(int q = -(lower_bound); q < (lower_bound+1); ++q){	 // z stencil
            for(int l = -(lower_bound); l < (lower_bound+1); ++l){   // x stencil
                for(int w = -(lower_bound); w < (lower_bound+1); ++w){	// y stencil

                    if((x_index + l) >= 0 && (x_index + l) < _max_x){
                        if((_z_index + q) >= 0 && (_z_index + q) < _max_z){
                            if((y + w) >= 0 && (y + w) < _max_y){

                                temp_index = (x_index + l)*_max_y + (((_z_index+q+ _stencil_size) % _stencil_size)*_max_y*_max_x) +y+w ;
                                neighbour_sum += (_stencil[counter]*_temp_vec[temp_index]);
                                //
                                counter++;

                            }
                        }
                    }


                }
            }
        }

        _pdata[global_index] = std::round(neighbour_sum/(pow(_stencil_size,3)*1.0));


    }
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    // Read provided APR file
    cmdLineOptions options = read_command_line_options(argc, argv);
    const int reps = 20;

    std::string fileName = options.directory + options.input;
    APR<uint16_t> apr;
    apr.read_apr(fileName);

    // Get dense representation of APR
    APRIterator<uint16_t> aprIt(apr);

    ///////////////////////////
    ///
    /// Sparse Data for GPU
    ///
    ///////////////////////////

    std::vector<std::tuple<std::size_t,std::size_t>> level_zx_index_start;//size = number of rows on all levels
    std::vector<std::uint16_t> y_explicit;y_explicit.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::uint16_t> particle_values;particle_values.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::size_t> level_offset(aprIt.level_max()+1,UINT64_MAX);//size = number of levels
    const int stencil_half = 2;
    const int stencil_size = 2*stencil_half+1;
    std::vector<std::float_t> stencil;		// the stencil on the host
    std::float_t stencil_value = 1;
    stencil.resize(pow(stencil_half*2 + 1,stencil_size),stencil_value);

    std::cout << stencil[0] << std::endl;


    std::size_t x = 0;
    std::size_t z = 0;

    std::size_t zx_counter = 0;
    std::size_t pcounter = 0;


    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        level_offset[level] = zx_counter;

        for (z = 0; z < aprIt.spatial_index_z_max(level); ++z) {
            for (x = 0; x < aprIt.spatial_index_x_max(level); ++x) {

                zx_counter++;
                if (aprIt.set_new_lzx(level, z, x) < UINT64_MAX) {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>(aprIt.global_index(),
                                                                                               aprIt.particles_zx_end(level,z,x))); //This stores the begining and end global index for each level_xz_row
                } else {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>((std::size_t)pcounter,(std::size_t) pcounter)); //This stores the begining and end global index for each level_
                }

                for (aprIt.set_new_lzx(level, z, x);
                     aprIt.global_index() < aprIt.particles_zx_end(level, z,
                                                                   x); aprIt.set_iterator_to_particle_next_particle()) {
                    y_explicit.emplace_back(aprIt.y());
                    particle_values.emplace_back(apr.particles_intensities[aprIt]);
                    pcounter++;

                }
            }

        }
    }


    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    ////////////////////
    ///
    /// Example of doing our level,z,x access using the GPU data structure
    ///
    /////////////////////
    auto start = std::chrono::high_resolution_clock::now();


    thrust::host_vector<thrust::tuple<std::size_t,std::size_t> > h_level_zx_index_start(level_zx_index_start.size());
    thrust::transform(level_zx_index_start.begin(), level_zx_index_start.end(),
                      h_level_zx_index_start.begin(),
                      [] ( const auto& _el ){
                          return thrust::make_tuple(std::get<0>(_el), std::get<1>(_el));
                      } );

    thrust::device_vector<thrust::tuple<std::size_t,std::size_t> > d_level_zx_index_start = h_level_zx_index_start;


    thrust::device_vector<std::float_t> d_stencil(stencil.begin(), stencil.end());		// device stencil
    thrust::device_vector<std::uint16_t> d_y_explicit(y_explicit.begin(), y_explicit.end());
    thrust::device_vector<std::uint16_t> d_particle_values(particle_values.begin(), particle_values.end());
    thrust::device_vector<std::uint16_t> d_test_access_data(d_particle_values.size(),0);

    thrust::device_vector<std::size_t> d_level_offset(level_offset.begin(),level_offset.end());


    std::size_t number_blocks = 8000;

    thrust::device_vector<std::uint16_t> d_x_end(number_blocks,0);
    std::uint16_t*   _x_end  =  thrust::raw_pointer_cast(d_x_end.data());

    thrust::device_vector<std::size_t> d_ind_end(number_blocks,0);
    std::size_t*   _ind_end  =  thrust::raw_pointer_cast(d_ind_end.data());

    std::size_t max_elements = 0;

    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        auto xtimesy = aprIt.spatial_index_y_max(level);// + (stencil_size - 1);
        xtimesy *= aprIt.spatial_index_x_max(level);// + (stencil_size - 1);
        if(max_elements < xtimesy)
            max_elements = xtimesy;
    }
    thrust::device_vector<std::uint16_t> d_temp_vec(max_elements*stencil_size,0);

    const thrust::tuple<std::size_t,std::size_t>* levels =  thrust::raw_pointer_cast(d_level_zx_index_start.data());
    const std::uint16_t*             y_ex   =  thrust::raw_pointer_cast(d_y_explicit.data());
    const std::uint16_t*             pdata  =  thrust::raw_pointer_cast(d_particle_values.data());
    const std::size_t*             offsets= thrust::raw_pointer_cast(d_level_offset.data());
    std::uint16_t*                   tvec = thrust::raw_pointer_cast(d_temp_vec.data());
    std::uint16_t*                   expected = thrust::raw_pointer_cast(d_test_access_data.data());
    const std::float_t*		     stencil_pointer =  thrust::raw_pointer_cast(d_stencil.data());		// stencil pointer


    if(hipGetLastError()!=hipSuccess){
        std::cerr << "memory transfers failed!\n";
    }
    auto end_gpu_tx = std::chrono::high_resolution_clock::now();

    for ( int r = 0;r<reps;++r){

        auto start_gpu_kernel = std::chrono::high_resolution_clock::now();

        for (int lvl = aprIt.level_min(); lvl <= aprIt.level_max(); ++lvl) {

            const int y_num = aprIt.spatial_index_y_max(lvl);
            const int x_num = aprIt.spatial_index_x_max(lvl);
            const int z_num = aprIt.spatial_index_z_max(lvl);

            dim3 threads(32);
            dim3 blocks((x_num + threads.x - 1) / threads.x);

//            for (int i = 0; i < stencil_half; ++i) {
//                insert<<<blocks,threads>>>(lvl,
//                        i,
//                        levels,
//                        y_ex,
//                        pdata,
//                        offsets,
//                        y_num,x_num,
//                        particle_values.size(),
//                        tvec,
//                        stencil_size, stencil_half);
//            }

            if (lvl <= aprIt.level_max()) {
                for (int z = 0; z < (z_num); ++z) {

                    if (aprIt.particles_z_begin(lvl, z) != UINT64_MAX) {

                        std::size_t num_parts = (aprIt.particles_z_end(lvl, z) - aprIt.particles_z_begin(lvl, z));
                        std::size_t num_blocks = std::min((std::size_t) number_blocks, num_parts);
                        std::float_t parts_per_block = num_parts / (num_blocks * 1.0f);
                        std::size_t parts_begin = aprIt.particles_z_begin(lvl, z);

                        unsigned int threads_d = 64;
                        unsigned int chunk = ceil((num_blocks) / (1.0f * threads_d))+1;

                        dim3 threads_load(threads_d);
                        dim3 blocks_load(chunk);

                        //std::cout << parts_per_block << std::endl;
//                        std::cout << parts_begin << std::endl;
//                        std::cout << num_parts << std::endl;


                        load_balance <<< blocks, threads >>>
                                                  (lvl, z, levels, _x_end, _ind_end, offsets, x_num, num_blocks+1, parts_per_block, parts_begin);
                        hipDeviceSynchronize();

                        //std::cout << num_blocks << " " << chunk << std::endl;

                        //test_iterate << < blocks_load, threads_load >> >
                                 //                      (lvl, z, levels, _x_end, _ind_end, offsets, x_num, expected, num_blocks+1);


                        insert_dynamic <<< blocks_load, threads_load >>> (lvl,
                                z,
                                levels,
                                y_ex,
                                pdata,
                                offsets,
                                y_num, x_num,
                                particle_values.size(),
                                tvec,
                                stencil_size, stencil_half,_x_end,_ind_end,num_blocks+1);


//                        insert << < blocks, threads >> > (lvl,
//                                z ,
//                                levels,
//                                y_ex,
//                                pdata,
//                                offsets,
//                                y_num, x_num,
//                                particle_values.size(),
//                                tvec,
//                                stencil_size, stencil_half);
//
                    if (hipGetLastError() != hipSuccess) {
                        std::cerr << "on " << lvl << " the cuda kernel does not run!\n";
                        break;
                    }
                    hipDeviceSynchronize();

//                    push_back << < blocks, threads >> > (lvl,
//                            z,
//                            levels,
//                            y_ex,
//                            tvec,
//                            offsets,
//                            y_num, x_num, z_num,
//                            particle_values.size(),
//                            expected,
//                            stencil_size, stencil_half, stencil_pointer);

                        push_back_dynamic <<< blocks_load, threads_load >>> (lvl,
                                z,
                                levels,
                                y_ex,
                                tvec,
                                offsets,
                                y_num, x_num, z_num,
                                particle_values.size(),
                                expected,
                                stencil_size, stencil_half, stencil_pointer,_x_end,_ind_end,num_blocks+1);

                    hipDeviceSynchronize();

                    }
                }
            }

        }
        hipDeviceSynchronize();
        auto end_gpu_kernel = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double,std::milli> rep_diff = end_gpu_kernel - start_gpu_kernel;
        std::cout << std::setw(3) << r << " GPU:      " << rep_diff  .count() << " ms\n";

    }

    auto end_gpu_kernels = std::chrono::high_resolution_clock::now();

    std::vector<std::uint16_t> test_access_data(d_test_access_data.size(),std::numeric_limits<std::uint16_t>::max());
    thrust::copy(d_test_access_data.begin(), d_test_access_data.end(), test_access_data.begin());

    auto end_gpu = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double,std::milli> gpu_tx_up = end_gpu_tx - start;
    std::chrono::duration<double,std::milli> gpu_tx_down = end_gpu - end_gpu_kernels;

    std::cout << "   GPU: up   " << gpu_tx_up  .count() << " ms\n";
    std::cout << "   GPU: down " << gpu_tx_down.count() << " ms\n";

    // assert(test_access_data.back() != std::numeric_limits<std::uint16_t>::max());

    //////////////////////////
    ///
    /// Now check the data
    ///
    ////////////////////////////



    ExtraParticleData<float> utest_data(apr);
    apr.parameters.input_dir = options.directory;


    create_test_particles_surya(apr,aprIt, utest_data,apr.particles_intensities,stencil, stencil_size, stencil_half);

    bool success = true;

    uint64_t c_fail= 0;



    for (uint64_t particle_number = 0; particle_number < apr.total_number_particles(); ++particle_number) {
        //This step is required for all loops to set the iterator by the particle number
        aprIt.set_iterator_to_particle_by_number(particle_number);


        if(utest_data.data[particle_number]!=test_access_data[particle_number]){
            success = false;

                //if(aprIt.level() == 6) {
//                    std::cout << particle_number << std::endl;
//                std::cout << aprIt.x() << " " << aprIt.y() << " " << aprIt.z() << " " << aprIt.level() << " expected: "
//                          << utest_data.data[particle_number] << ", received: " << test_access_data[particle_number]
//                          << std::endl;
            //}
            //break;
            c_fail++;
        }

        // std::cout << aprIt.x()<< " "  << aprIt.y()<< " "  << aprIt.z() << " "<< aprIt.level() << " expected: " << utest_data.data[particle_number] << ", received: " << test_access_data[particle_number] << "\n";

    }


    if(success){
        std::cout << "PASS" << std::endl;
    } else {
        std::cout << "FAIL " << c_fail << std::endl;
    }


}



void create_test_particles_surya(APR<uint16_t>& apr,APRIterator<uint16_t>& apr_iterator,ExtraParticleData<float> &test_particles,ExtraParticleData<uint16_t>& particles,std::vector<float>& stencil, const int stencil_size, const int stencil_half){

    for (uint64_t level_local = apr_iterator.level_max(); level_local >= apr_iterator.level_min(); --level_local) {


        MeshData<float> by_level_recon;
        by_level_recon.init(apr_iterator.spatial_index_y_max(level_local),apr_iterator.spatial_index_x_max(level_local),apr_iterator.spatial_index_z_max(level_local),0);

        uint64_t level = level_local;

        const int step_size = 1;

        uint64_t particle_number;

        for (particle_number = apr_iterator.particles_level_begin(level);
             particle_number < apr_iterator.particles_level_end(level); ++particle_number) {
            //
            //  Parallel loop over level
            //
            apr_iterator.set_iterator_to_particle_by_number(particle_number);

            int dim1 = apr_iterator.y() ;
            int dim2 = apr_iterator.x() ;
            int dim3 = apr_iterator.z() ;

            float temp_int;
            //add to all the required rays

            temp_int = particles[apr_iterator];

            const int offset_max_dim1 = std::min((int) by_level_recon.y_num, (int) (dim1 + step_size));
            const int offset_max_dim2 = std::min((int) by_level_recon.x_num, (int) (dim2 + step_size));
            const int offset_max_dim3 = std::min((int) by_level_recon.z_num, (int) (dim3 + step_size));

            for (int64_t q = dim3; q < offset_max_dim3; ++q) {

                for (int64_t k = dim2; k < offset_max_dim2; ++k) {
                    for (int64_t i = dim1; i < offset_max_dim1; ++i) {
                        by_level_recon.mesh[i + (k) * by_level_recon.y_num + q * by_level_recon.y_num * by_level_recon.x_num] = temp_int;
                    }
                }
            }
        }


        int x = 0;
        int z = 0;


        for (z = 0; z < (apr.spatial_index_z_max(level)); ++z) {
            //lastly loop over particle locations and compute filter.
            for (x = 0; x < apr.spatial_index_x_max(level); ++x) {
                for (apr_iterator.set_new_lzx(level, z, x);
                     apr_iterator.global_index() < apr_iterator.particles_zx_end(level, z,
                                                                                 x); apr_iterator.set_iterator_to_particle_next_particle()) {
                    double neigh_sum = 0;
                    float counter = 0;

                    const int k = apr_iterator.y(); // offset to allow for boundary padding
                    const int i = x;

                    //test_particles[apr_iterator]=0;

                    for (int l = -stencil_half; l < stencil_half+1; ++l) {
                        for (int q = -stencil_half; q < stencil_half+1; ++q) {
                            for (int w = -stencil_half; w < stencil_half+1; ++w) {

                                if((k+w)>=0 & (k+w) < (apr.spatial_index_y_max(level))){
                                    if((i+q)>=0 & (i+q) < (apr.spatial_index_x_max(level))){
                                        if((z+l)>=0 & (z+l) < (apr.spatial_index_z_max(level))){
                                            neigh_sum += stencil[counter] * by_level_recon.at(k + w, i + q, z+l);
                                            //neigh_sum += by_level_recon.at(k + w, i + q, z+l);
                                            //if(l==1) {
                                            //  test_particles[apr_iterator] = by_level_recon.at(k, i , z+l);
                                            //}
                                        }
                                    }
                                }
                                counter++;
                            }
                        }
                    }

                    test_particles[apr_iterator] = std::round(neigh_sum/(pow(stencil_size,3)*1.0));
                    test_particles[apr_iterator] = 1;

                }
            }
        }

        //std::string image_file_name = apr.parameters.input_dir + std::to_string(level_local) + "_by_level.tif";
       // TiffUtils::saveMeshAsTiff(image_file_name, by_level_recon);
    }

}


